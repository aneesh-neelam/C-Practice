#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void square(float * d_out, float * d_in)
{
	int idx = threadIdx.x; // Thread Index within the block
	float f = d_in[idx];
	d_out[idx] = f * f;
}

int main(int argc, char ** argv) 
{
	const int ARRAY_SIZE = 96;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) 
	{
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	square<<<1, ARRAY_SIZE>>>(d_out, d_in);
	// Syntax: Identifier<<<Blocks, Thread per block>>>(params...)
	// Thread Limit: 1024 per block (or 512 for older Nvidia GPUs)
	// Dimentionality using dim3(x,y,z)
	// Identifier<<<dim3(bx,by,bz),dim3(tx,ty,tz),shmem>>>(params...)

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < ARRAY_SIZE; i++) 
	{
		printf("%f\n", h_out[i]);
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}

